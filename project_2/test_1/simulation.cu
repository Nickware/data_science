#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <cstring>  // Para strcpy
#include <hip/hip_runtime.h>

// Constantes
const int N = 50;              // 50,000 partículas
const int FRAMES = 1;           // 100 frames de simulación
const float UMBRAL_VELOCIDAD = 0.5f; // Demonio: solo partículas rápidas pasan

// Estructura de una partícula (CPU y GPU)
struct Particle {
    float x, y;     // Posición
    float vx, vy;   // Velocidad
    int color;      // 0: verde (inscrito), 1: rojo (no inscrito)
    char numero[5]; // Número de identificación (ej: "1234")
};

// Kernel CUDA: actualizar posiciones y aplicar "demonio"
__global__ void actualizarParticulas(Particle* particulas, int frame_actual) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Mover partícula (dinámica simplificada)
    particulas[idx].x += particulas[idx].vx * 0.001f;
    particulas[idx].y += particulas[idx].vy * 0.001f;

    // Rebote en bordes de la caja (0 < x < 1, 0 < y < 1)
    if (particulas[idx].x <= 0 || particulas[idx].x >= 1) particulas[idx].vx *= -1;
    if (particulas[idx].y <= 0 || particulas[idx].y >= 1) particulas[idx].vy *= -1;

    // Demonio de Maxwell: compuerta en x=0.5
    if (particulas[idx].x < 0.5f && particulas[idx].vx > UMBRAL_VELOCIDAD) {
        particulas[idx].x += 0.01f;  // Permitir paso a la derecha
    }
}

// Función para guardar un frame en formato GNUPLOT
void guardarFrame(const Particle* particulas, int frame_id) {
    std::ofstream archivo("datos/frame_" + std::to_string(frame_id) + ".dat");
    for (int i = 0; i < N; ++i) {
        archivo << particulas[i].x << " " << particulas[i].y << " " 
                << particulas[i].color << " " << particulas[i].numero << "\n";
    }
    archivo.close();
}

int main() {
    // Reservar memoria en GPU
    Particle* d_particulas;
    hipMalloc(&d_particulas, N * sizeof(Particle));

    // Inicializar partículas en CPU (50% verdes "inscritas", 50% rojas)
    Particle* particulas = new Particle[N];
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> pos_dist(0.0f, 1.0f);
    std::uniform_real_distribution<float> vel_dist(-1.0f, 1.0f);

    for (int i = 0; i < N; ++i) {
        particulas[i].x = pos_dist(gen);
        particulas[i].y = pos_dist(gen);
        particulas[i].vx = vel_dist(gen);
        particulas[i].vy = vel_dist(gen);
        particulas[i].color = (i < N/2) ? 0 : 1;  // Primeras 25k verdes, resto rojas
        
        // Asignar número aleatorio de 4 dígitos a las verdes
        if (particulas[i].color == 0) {
            snprintf(particulas[i].numero, 5, "%04d", std::uniform_int_distribution<int>(1000, 9999)(gen));
        } else {
            strcpy(particulas[i].numero, "NA");
        }
    }

    // Copiar datos a GPU
    hipMemcpy(d_particulas, particulas, N * sizeof(Particle), hipMemcpyHostToDevice);

    // Simular y guardar frames
    for (int frame = 0; frame < FRAMES; ++frame) {
        // Lanzar kernel en GPU (256 hilos por bloque)
        dim3 blockSize(256);
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
        actualizarParticulas<<<gridSize, blockSize>>>(d_particulas, frame);
        
        // Copiar datos de vuelta a CPU para guardar
        hipMemcpy(particulas, d_particulas, N * sizeof(Particle), hipMemcpyDeviceToHost);
        
        // Guardar frame actual
        guardarFrame(particulas, frame);
    }

    // Liberar memoria
    delete[] particulas;
    hipFree(d_particulas);

    std::cout << "Simulación completada. Datos guardados en /datos/\n";
    return 0;
}