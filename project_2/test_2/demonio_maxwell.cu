#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <cstring>
#include <sys/stat.h> // Para mkdir
#include <hip/hip_runtime.h>

// Constantes
const int N = 10;                   // 10 partículas (5 verdes, 5 rojas)
const int FRAMES = 500;             // Más frames para ver evolución
const float UMBRAL_VELOCIDAD = 0.7f; // Demonio: solo partículas rojas rápidas pasan

struct Particle {
    float x, y;     // Posición (0 < x < 2, 0 < y < 1)
    float vx, vy;   // Velocidad
    int color;      // 0: verde (no pasa), 1: rojo (puede pasar)
};

__global__ void actualizarParticulas(Particle* particulas, int frame_actual) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Mover partícula (dinámica más rápida)
    particulas[idx].x += particulas[idx].vx * 0.02f;
    particulas[idx].y += particulas[idx].vy * 0.02f;

    // Rebote en bordes (caja rectangular: 0 < x < 2, 0 < y < 1)
    if (particulas[idx].x <= 0 || particulas[idx].x >= 2) particulas[idx].vx *= -1;
    if (particulas[idx].y <= 0 || particulas[idx].y >= 1) particulas[idx].vy *= -1;

    // Demonio de Maxwell: compuerta en x=1 (solo rojas rápidas pasan a la derecha)
    if (particulas[idx].x < 1.0f && particulas[idx].vx > UMBRAL_VELOCIDAD && particulas[idx].color == 1) {
        particulas[idx].x += 0.05f;  // Pasar a la derecha
    }
}

void guardarFrame(const Particle* particulas, int frame_id) {
    std::ofstream archivo("datos/frame_" + std::to_string(frame_id) + ".dat");
    // Dibujar compuerta (línea vertical en x=1)
    archivo << "1 0\n1 1\n\n";  // Formato para línea en GNUPLOT
    
    // Guardar partículas (verdes: círculos, rojas: cruces)
    for (int i = 0; i < N; ++i) {
        archivo << particulas[i].x << " " << particulas[i].y << " " << particulas[i].color << "\n";
    }
    archivo.close();
}

int main() {
    // Crear carpeta "datos" si no existe
    mkdir("datos", 0777);

    Particle* d_particulas;
    hipMalloc(&d_particulas, N * sizeof(Particle));

    Particle particulas[N];
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> pos_x(0.0f, 1.0f);  // Inicialmente todas a la izquierda
    std::uniform_real_distribution<float> pos_y(0.0f, 1.0f);
    std::uniform_real_distribution<float> vel(-1.5f, 1.5f);   // Velocidades más altas

    // Inicializar 5 verdes (no pasan) y 5 rojas (pueden pasar)
    for (int i = 0; i < N; ++i) {
        particulas[i].x = pos_x(gen);
        particulas[i].y = pos_y(gen);
        particulas[i].vx = vel(gen);
        particulas[i].vy = vel(gen);
        particulas[i].color = (i < 5) ? 0 : 1;  // Primeras 5 verdes, resto rojas
    }

    hipMemcpy(d_particulas, particulas, N * sizeof(Particle), hipMemcpyHostToDevice);

    for (int frame = 0; frame < FRAMES; ++frame) {
        dim3 blockSize(256);
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
        actualizarParticulas<<<gridSize, blockSize>>>(d_particulas, frame);
        hipMemcpy(particulas, d_particulas, N * sizeof(Particle), hipMemcpyDeviceToHost);
        guardarFrame(particulas, frame);
    }

    hipFree(d_particulas);
    std::cout << "Simulación completada. Ejecuta: ./generar_video.sh\n";
    return 0;
}
